#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <float.h>
#include <math.h>
#include <time.h>

#include "../../constants.h"

#define N_RADIUS 4
#define N_THREADS_PER_PLANE_DIM_X 16
#define N_THREADS_PER_PLANE_DIM_Y 16

__device__ const int z_idx[] = {0,1,2,3,4,5,6,7,8,0,1,2,3,4,5,6,7,8};

__global__ void target_inner_3d_kernel(
    llint nx, llint ny, llint nz,
    llint x3, llint x4, llint y3, llint y4, llint z3, llint z4,
    llint lx, llint ly, llint lz,
    float hdx_2, float hdy_2, float hdz_2,
    float coef0,
    float coefx_1, float coefx_2, float coefx_3, float coefx_4,
    float coefy_1, float coefy_2, float coefy_3, float coefy_4,
    float coefz_1, float coefz_2, float coefz_3, float coefz_4,
    const float *__restrict__ u, float *__restrict__ v, const float *__restrict__ vp,
    const float *__restrict__ phi, const float *__restrict__ eta
) {
    const int nz_s_u = 1+2*N_RADIUS;
    __shared__ float s_u[nz_s_u][N_THREADS_PER_PLANE_DIM_Y+2*N_RADIUS][N_THREADS_PER_PLANE_DIM_X+2*N_RADIUS];

    const llint j0 = y3 + blockIdx.y * blockDim.y;
    const llint k0 = z3 + blockIdx.x * blockDim.x;

    const llint je = min(j0 + N_THREADS_PER_PLANE_DIM_Y, y4);
    const llint ke = min(k0 + N_THREADS_PER_PLANE_DIM_X, z4);

    const llint j = j0 + threadIdx.y;
    const llint k = k0 + threadIdx.x;

    const llint sje = (j0+N_THREADS_PER_PLANE_DIM_Y<y4) ? N_THREADS_PER_PLANE_DIM_Y : ((y4-y3-1)%N_THREADS_PER_PLANE_DIM_Y+1);
    const llint ske = (k0+N_THREADS_PER_PLANE_DIM_X<z4) ? N_THREADS_PER_PLANE_DIM_X : ((z4-z3-1)%N_THREADS_PER_PLANE_DIM_X+1);

    const llint suj = threadIdx.y + N_RADIUS;
    const llint suk = threadIdx.x + N_RADIUS;

    // Preparation
    for (llint i = x3-N_RADIUS; i < x3+N_RADIUS; i++) {
        int z = (i + nz_s_u) % nz_s_u;
        s_u[z][suj][suk] = u[IDX3_l(i,j,k)];
        if (threadIdx.y < N_RADIUS) {
            s_u[z][threadIdx.y][suk] = u[IDX3_l(i, j - N_RADIUS, k)];
            s_u[z][threadIdx.y+sje+N_RADIUS][suk] = u[IDX3_l(i, threadIdx.y+je, k)];
        }
        if (threadIdx.x < N_RADIUS) {
            s_u[z][suj][threadIdx.x] = u[IDX3_l(i,j,k - N_RADIUS)];
            s_u[z][suj][threadIdx.x+ske+N_RADIUS] = u[IDX3_l(i,j,threadIdx.x+ke)];
        }
    }

    // Streaming
    for (llint i = x3; i < x4; i++) {
        int z = (i + nz_s_u) % nz_s_u;
        int z_R = z_idx[z + N_RADIUS];

        s_u[z_R][suj][suk] = u[IDX3_l(i+N_RADIUS,j,k)];
        if (threadIdx.y < N_RADIUS) {
            s_u[z_R][threadIdx.y][suk] = u[IDX3_l(i+N_RADIUS, j - N_RADIUS, k)];
            s_u[z_R][threadIdx.y+sje+N_RADIUS][suk] = u[IDX3_l(i+N_RADIUS, threadIdx.y+je, k)];
        }
        if (threadIdx.x < N_RADIUS) {
            s_u[z_R][suj][threadIdx.x] = u[IDX3_l(i+N_RADIUS,j,k - N_RADIUS)];
            s_u[z_R][suj][threadIdx.x+ske+N_RADIUS] = u[IDX3_l(i+N_RADIUS,j,threadIdx.x+ke)];
        }
        __syncthreads();

        if (j < y4 && k < z4) {
            float lap = __fmaf_rn(coef0, s_u[z][suj][suk]
                    , __fmaf_rn(coefx_1, __fadd_rn(s_u[z_idx[z+1]][suj][suk],s_u[z_idx[z-1+nz_s_u]][suj][suk])
                    , __fmaf_rn(coefy_1, __fadd_rn(s_u[z][suj+1][suk],s_u[z][suj-1][suk])
                    , __fmaf_rn(coefz_1, __fadd_rn(s_u[z][suj][suk+1],s_u[z][suj][suk-1])
                    , __fmaf_rn(coefx_2, __fadd_rn(s_u[z_idx[z+2]][suj][suk],s_u[z_idx[z-2+nz_s_u]][suj][suk])
                    , __fmaf_rn(coefy_2, __fadd_rn(s_u[z][suj+2][suk],s_u[z][suj-2][suk])
                    , __fmaf_rn(coefz_2, __fadd_rn(s_u[z][suj][suk+2],s_u[z][suj][suk-2])
                    , __fmaf_rn(coefx_3, __fadd_rn(s_u[z_idx[z+3]][suj][suk],s_u[z_idx[z-3+nz_s_u]][suj][suk])
                    , __fmaf_rn(coefy_3, __fadd_rn(s_u[z][suj+3][suk],s_u[z][suj-3][suk])
                    , __fmaf_rn(coefz_3, __fadd_rn(s_u[z][suj][suk+3],s_u[z][suj][suk-3])
                    , __fmaf_rn(coefx_4, __fadd_rn(s_u[z_idx[z+4]][suj][suk],s_u[z_idx[z-4+nz_s_u]][suj][suk])
                    , __fmaf_rn(coefy_4, __fadd_rn(s_u[z][suj+4][suk],s_u[z][suj-4][suk])
                    , __fmul_rn(coefz_4, __fadd_rn(s_u[z][suj][suk+4],s_u[z][suj][suk-4])
            )))))))))))));

            v[IDX3_l(i,j,k)] = __fmaf_rn(2.f, s_u[z][suj][suk],
                __fmaf_rn(vp[IDX3(i,j,k)], lap, -v[IDX3_l(i,j,k)])
            );
        }
    }
}

__global__ void target_pml_3d_kernel(
    llint nx, llint ny, llint nz,
    llint x3, llint x4, llint y3, llint y4, llint z3, llint z4,
    llint lx, llint ly, llint lz,
    float hdx_2, float hdy_2, float hdz_2,
    float coef0,
    float coefx_1, float coefx_2, float coefx_3, float coefx_4,
    float coefy_1, float coefy_2, float coefy_3, float coefy_4,
    float coefz_1, float coefz_2, float coefz_3, float coefz_4,
    const float *__restrict__ u, float *__restrict__ v, const float *__restrict__ vp,
    float *__restrict__ phi, const float *__restrict__ eta
) {
    const int nz_s_u = 1+2*N_RADIUS;
    __shared__ float s_u[nz_s_u][N_THREADS_PER_PLANE_DIM_Y+2*N_RADIUS][N_THREADS_PER_PLANE_DIM_X+2*N_RADIUS];

    const llint j0 = y3 + blockIdx.y * blockDim.y;
    const llint k0 = z3 + blockIdx.x * blockDim.x;

    const llint je = min(j0 + N_THREADS_PER_PLANE_DIM_Y, y4);
    const llint ke = min(k0 + N_THREADS_PER_PLANE_DIM_X, z4);

    const llint j = j0 + threadIdx.y;
    const llint k = k0 + threadIdx.x;

    const llint sje = (j0+N_THREADS_PER_PLANE_DIM_Y<y4) ? N_THREADS_PER_PLANE_DIM_Y : ((y4-y3-1)%N_THREADS_PER_PLANE_DIM_Y+1);
    const llint ske = (k0+N_THREADS_PER_PLANE_DIM_X<z4) ? N_THREADS_PER_PLANE_DIM_X : ((z4-z3-1)%N_THREADS_PER_PLANE_DIM_X+1);

    const llint suj = threadIdx.y + N_RADIUS;
    const llint suk = threadIdx.x + N_RADIUS;

    // Preparation
    for (llint i = x3-N_RADIUS; i < x3+N_RADIUS; i++) {
        int z = (i + nz_s_u) % nz_s_u;
        s_u[z][suj][suk] = u[IDX3_l(i,j,k)];
        if (threadIdx.y < N_RADIUS) {
            s_u[z][threadIdx.y][suk] = u[IDX3_l(i, j - N_RADIUS, k)];
            s_u[z][threadIdx.y+sje+N_RADIUS][suk] = u[IDX3_l(i, threadIdx.y+je, k)];
        }
        if (threadIdx.x < N_RADIUS) {
            s_u[z][suj][threadIdx.x] = u[IDX3_l(i,j,k - N_RADIUS)];
            s_u[z][suj][threadIdx.x+ske+N_RADIUS] = u[IDX3_l(i,j,threadIdx.x+ke)];
        }
    }

    // Streaming
    for (llint i = x3; i < x4; i++) {
        int z = (i + nz_s_u) % nz_s_u;
        int z_R = z_idx[z + N_RADIUS];

        s_u[z_R][suj][suk] = u[IDX3_l(i+N_RADIUS,j,k)];
        if (threadIdx.y < N_RADIUS) {
            s_u[z_R][threadIdx.y][suk] = u[IDX3_l(i+N_RADIUS, j - N_RADIUS, k)];
            s_u[z_R][threadIdx.y+sje+N_RADIUS][suk] = u[IDX3_l(i+N_RADIUS, threadIdx.y+je, k)];
        }
        if (threadIdx.x < N_RADIUS) {
            s_u[z_R][suj][threadIdx.x] = u[IDX3_l(i+N_RADIUS,j,k - N_RADIUS)];
            s_u[z_R][suj][threadIdx.x+ske+N_RADIUS] = u[IDX3_l(i+N_RADIUS,j,threadIdx.x+ke)];
        }
        __syncthreads();

        if (j < y4 && k < z4) {
            float lap = __fmaf_rn(coef0, s_u[z][suj][suk]
                    , __fmaf_rn(coefx_1, __fadd_rn(s_u[z_idx[z+1]][suj][suk],s_u[z_idx[z-1+nz_s_u]][suj][suk])
                    , __fmaf_rn(coefy_1, __fadd_rn(s_u[z][suj+1][suk],s_u[z][suj-1][suk])
                    , __fmaf_rn(coefz_1, __fadd_rn(s_u[z][suj][suk+1],s_u[z][suj][suk-1])
                    , __fmaf_rn(coefx_2, __fadd_rn(s_u[z_idx[z+2]][suj][suk],s_u[z_idx[z-2+nz_s_u]][suj][suk])
                    , __fmaf_rn(coefy_2, __fadd_rn(s_u[z][suj+2][suk],s_u[z][suj-2][suk])
                    , __fmaf_rn(coefz_2, __fadd_rn(s_u[z][suj][suk+2],s_u[z][suj][suk-2])
                    , __fmaf_rn(coefx_3, __fadd_rn(s_u[z_idx[z+3]][suj][suk],s_u[z_idx[z-3+nz_s_u]][suj][suk])
                    , __fmaf_rn(coefy_3, __fadd_rn(s_u[z][suj+3][suk],s_u[z][suj-3][suk])
                    , __fmaf_rn(coefz_3, __fadd_rn(s_u[z][suj][suk+3],s_u[z][suj][suk-3])
                    , __fmaf_rn(coefx_4, __fadd_rn(s_u[z_idx[z+4]][suj][suk],s_u[z_idx[z-4+nz_s_u]][suj][suk])
                    , __fmaf_rn(coefy_4, __fadd_rn(s_u[z][suj+4][suk],s_u[z][suj-4][suk])
                    , __fmul_rn(coefz_4, __fadd_rn(s_u[z][suj][suk+4],s_u[z][suj][suk-4])
            )))))))))))));

            const float s_eta_c = eta[IDX3_eta1(i,j,k)];

            v[IDX3_l(i,j,k)] = __fdiv_rn(
                __fmaf_rn(
                    __fmaf_rn(2.f, s_eta_c,
                        __fsub_rn(2.f,
                            __fmul_rn(s_eta_c, s_eta_c)
                        )
                    ),
                    s_u[z][suj][suk],
                    __fmaf_rn(
                        vp[IDX3(i,j,k)],
                        __fadd_rn(lap, phi[IDX3(i,j,k)]),
                        -v[IDX3_l(i,j,k)]
                    )
                ),
                __fmaf_rn(2.f, s_eta_c, 1.f)
            );

            phi[IDX3(i,j,k)] = __fdiv_rn(
                    __fsub_rn(
                        phi[IDX3(i,j,k)],
                        __fmaf_rn(
                        __fmul_rn(
                            __fsub_rn(eta[IDX3_eta1(i+1,j,k)], eta[IDX3_eta1(i-1,j,k)]),
                            __fsub_rn(s_u[z_idx[z+1]][suj][suk], s_u[z_idx[z-1+nz_s_u]][suj][suk])
                        ), hdx_2,
                        __fmaf_rn(
                        __fmul_rn(
                            __fsub_rn(eta[IDX3_eta1(i,j+1,k)], eta[IDX3_eta1(i,j-1,k)]),
                            __fsub_rn(s_u[z][suj+1][suk], s_u[z][suj-1][suk])
                        ), hdy_2,
                        __fmul_rn(
                            __fmul_rn(
                                __fsub_rn(eta[IDX3_eta1(i,j,k+1)], eta[IDX3_eta1(i,j,k-1)]),
                                __fsub_rn(s_u[z][suj][suk+1], s_u[z][suj][suk-1])
                            ),
                        hdz_2)
                        ))
                    )
                ,
                __fadd_rn(1.f, s_eta_c)
            );
        }
    }
}

__global__ void kernel_add_source_kernel(float *g_u, llint idx, float source) {
    g_u[idx] += source;
}

extern "C" void target(
    uint nsteps, double *time_kernel,
    llint nx, llint ny, llint nz,
    llint x1, llint x2, llint x3, llint x4, llint x5, llint x6,
    llint y1, llint y2, llint y3, llint y4, llint y5, llint y6,
    llint z1, llint z2, llint z3, llint z4, llint z5, llint z6,
    llint lx, llint ly, llint lz,
    llint sx, llint sy, llint sz,
    float hdx_2, float hdy_2, float hdz_2,
    const float *__restrict__ coefx, const float *__restrict__ coefy, const float *__restrict__ coefz,
    float *__restrict__ u, const float *__restrict__ v, const float *__restrict__ vp,
    const float *__restrict__ phi, const float *__restrict__ eta, const float *__restrict__ source
) {
    struct timespec start, end;

    const llint size_u = (nx + 2 * lx) * (ny + 2 * ly) * (nz + 2 * lz);
    const llint size_v = size_u;
    const llint size_phi = nx*ny*nz;
    const llint size_vp = size_phi;
    const llint size_eta = (nx+2)*(ny+2)*(nz+2);

    const llint size_u_ext = (nx + 2 * lx)
                           * ((((ny+N_THREADS_PER_PLANE_DIM_Y-1) / N_THREADS_PER_PLANE_DIM_Y + 1) * N_THREADS_PER_PLANE_DIM_Y) + 2 * ly)
                           * ((((nz+N_THREADS_PER_PLANE_DIM_X-1) / N_THREADS_PER_PLANE_DIM_X + 1) * N_THREADS_PER_PLANE_DIM_X) + 2 * lz);

    float *d_u, *d_v, *d_vp, *d_phi, *d_eta;
    hipMalloc(&d_u, sizeof(float) * size_u_ext);
    hipMalloc(&d_v, sizeof(float) * size_u_ext);
    hipMalloc(&d_vp, sizeof(float) * size_vp);
    hipMalloc(&d_phi, sizeof(float) * size_phi);
    hipMalloc(&d_eta, sizeof(float) * size_eta);

    hipMemcpy(d_u, u, sizeof(float) * size_u, hipMemcpyHostToDevice);
    hipMemcpy(d_v, v, sizeof(float) * size_v, hipMemcpyHostToDevice);
    hipMemcpy(d_vp, vp, sizeof(float) * size_vp, hipMemcpyHostToDevice);
    hipMemcpy(d_phi, phi, sizeof(float) * size_phi, hipMemcpyHostToDevice);
    hipMemcpy(d_eta, eta, sizeof(float) * size_eta, hipMemcpyHostToDevice);

    const llint xmin = 0; const llint xmax = nx;
    const llint ymin = 0; const llint ymax = ny;

    dim3 threadsPerBlock(N_THREADS_PER_PLANE_DIM_X, N_THREADS_PER_PLANE_DIM_Y, 1);

    int num_streams = 7;
    hipStream_t streams[num_streams];
    for (int i = 0; i < num_streams; i++) {
        hipStreamCreateWithFlags(&(streams[i]), hipStreamNonBlocking);
    }

    const uint npo = 100;
    for (uint istep = 1; istep <= nsteps; ++istep) {
        clock_gettime(CLOCK_REALTIME, &start);

        dim3 n_block_front(
            (z2-z1+N_THREADS_PER_PLANE_DIM_X-1) / N_THREADS_PER_PLANE_DIM_X,
            (ny+N_THREADS_PER_PLANE_DIM_Y-1) / N_THREADS_PER_PLANE_DIM_Y);
        target_pml_3d_kernel<<<n_block_front, threadsPerBlock, 0, streams[1]>>>(nx,ny,nz,
            xmin,xmax,ymin,ymax,z1,z2,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_top(
            (z4-z3+N_THREADS_PER_PLANE_DIM_X-1) / N_THREADS_PER_PLANE_DIM_X,
            (y2-y1+N_THREADS_PER_PLANE_DIM_Y-1) / N_THREADS_PER_PLANE_DIM_Y);
        target_pml_3d_kernel<<<n_block_top, threadsPerBlock, 0, streams[2]>>>(nx,ny,nz,
            xmin,xmax,y1,y2,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_left(
            (z4-z3+N_THREADS_PER_PLANE_DIM_X-1) / N_THREADS_PER_PLANE_DIM_X,
            (y4-y3+N_THREADS_PER_PLANE_DIM_Y-1) / N_THREADS_PER_PLANE_DIM_Y,
            1);
        target_pml_3d_kernel<<<n_block_left, threadsPerBlock, 0, streams[3]>>>(nx,ny,nz,
            x1,x2,y3,y4,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_center(
            (z4-z3+N_THREADS_PER_PLANE_DIM_X-1) / N_THREADS_PER_PLANE_DIM_X,
            (y4-y3+N_THREADS_PER_PLANE_DIM_Y-1) / N_THREADS_PER_PLANE_DIM_Y);
        target_inner_3d_kernel<<<n_block_center, threadsPerBlock, 0, streams[0]>>>(nx,ny,nz,
            x3,x4,y3,y4,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_right(
            (z4-z3+N_THREADS_PER_PLANE_DIM_X-1) / N_THREADS_PER_PLANE_DIM_X,
            (y4-y3+N_THREADS_PER_PLANE_DIM_Y-1) / N_THREADS_PER_PLANE_DIM_Y,
            1);
        target_pml_3d_kernel<<<n_block_right, threadsPerBlock, 0, streams[4]>>>(nx,ny,nz,
            x5,x6,y3,y4,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_bottom(
            (z4-z3+N_THREADS_PER_PLANE_DIM_X-1) / N_THREADS_PER_PLANE_DIM_X,
            (y6-y5+N_THREADS_PER_PLANE_DIM_Y-1) / N_THREADS_PER_PLANE_DIM_Y,
            1);
        target_pml_3d_kernel<<<n_block_bottom, threadsPerBlock, 0, streams[5]>>>(nx,ny,nz,
            xmin,xmax,y5,y6,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_back(
            (z6-z5+N_THREADS_PER_PLANE_DIM_X-1) / N_THREADS_PER_PLANE_DIM_X,
            (ny+N_THREADS_PER_PLANE_DIM_Y-1) / N_THREADS_PER_PLANE_DIM_Y,
            1);
        target_pml_3d_kernel<<<n_block_back, threadsPerBlock, 0, streams[6]>>>(nx,ny,nz,
            xmin,xmax,ymin,ymax,z5,z6,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        for (int i = 0; i < num_streams; i++) {
            hipStreamSynchronize(streams[i]);
        }

        kernel_add_source_kernel<<<1, 1>>>(d_v, IDX3_l(sx,sy,sz), source[istep]);
        clock_gettime(CLOCK_REALTIME, &end);
        *time_kernel += (end.tv_sec  - start.tv_sec) +
                        (double)(end.tv_nsec - start.tv_nsec) / 1.0e9;

        float *t = d_u;
        d_u = d_v;
        d_v = t;

        // Print out
        if (istep % npo == 0) {
            printf("time step %u / %u\n", istep, nsteps);
        }
    }


    for (int i = 0; i < num_streams; i++) {
        hipStreamDestroy(streams[i]);
    }


    hipMemcpy(u, d_u, sizeof(float) * size_u, hipMemcpyDeviceToHost);

    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_vp);
    hipFree(d_phi);
    hipFree(d_eta);
}
