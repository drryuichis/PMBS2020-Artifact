#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <float.h>
#include <math.h>
#include <time.h>

#include "../../constants.h"

#define N_THREADS_PER_BLOCK 1024

__global__ void find_min_max_u_kernel(
    const float *__restrict__ g_u, float *__restrict__ g_max, float *__restrict__ g_min
) {
    extern __shared__ float sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int tidFromBack = blockDim.x - 1 - tid;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = g_u[i];
    __syncthreads();

    for (unsigned int s = blockDim.x/2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            if (sdata[tid + s] > sdata[tid])
            {
                sdata[tid] = sdata[tid + s];
            }
        }
        if (tidFromBack < s) {
            if (sdata[tid - s] < sdata[tid]) {
                sdata[tid] = sdata[tid - s];
            }
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        g_max[blockIdx.x] = sdata[0];
    }
    if (tidFromBack == 0)
    {
        g_min[blockIdx.x] = sdata[tid];
    }
}

extern "C" void find_min_max_u_cuda(
    const float *__restrict__ u, llint u_size, float *__restrict__ min_u, float *__restrict__ max_u
) {
    llint u_block = u_size / N_THREADS_PER_BLOCK;
    llint u_remainder = u_size % N_THREADS_PER_BLOCK;

    llint d_block = u_block;
    if (u_remainder != 0) { d_block += 1; }
    llint d_size = d_block * N_THREADS_PER_BLOCK;

    float *remainder = 0;
    llint remainder_size = 0;
    if (u_remainder != 0) {
        remainder_size = N_THREADS_PER_BLOCK - u_remainder;
        remainder = (float *)malloc(remainder_size * sizeof(float));
        memcpy(remainder, u, remainder_size * sizeof(float));
    }

    float* max = (float*)malloc(d_block * sizeof(float));
    float *min = (float*)malloc(d_block * sizeof(float));

    float* d_u, * d_max, * d_min;
    hipMalloc(&d_u, d_size * sizeof(float));
    hipMalloc(&d_max, d_block * sizeof(float));
    hipMalloc(&d_min, d_block * sizeof(float));

    hipMemcpy(d_u, u, u_size * sizeof(float), hipMemcpyHostToDevice);
    if (u_remainder != 0) {
        hipMemcpy(d_u+u_size, remainder, remainder_size * sizeof(float), hipMemcpyHostToDevice);
    }
    find_min_max_u_kernel<<<d_block, N_THREADS_PER_BLOCK, sizeof(float) * N_THREADS_PER_BLOCK>>>(d_u, d_max, d_min);
    hipMemcpy(max, d_max, d_block * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(min, d_min, d_block * sizeof(float), hipMemcpyDeviceToHost);

    *min_u = FLT_MAX, *max_u = FLT_MIN;
    for (size_t i = 0; i < d_block; i++) {
        *min_u = fminf(*min_u, min[i]);
        *max_u = fmaxf(*max_u, max[i]);
    }

    hipFree(d_max);
    hipFree(d_min);
    hipFree(d_u);
    free(remainder);
    free(max);
    free(min);
}
