#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <float.h>
#include <math.h>
#include <time.h>

#include "../../constants.h"

#define N_RADIUS 4
#define N_THREADS_PER_PLANE_DIM_X 16
#define N_THREADS_PER_PLANE_DIM_Y 16

__global__ void target_inner_3d_kernel(
    llint nx, llint ny, llint nz,
    llint x3, llint x4, llint y3, llint y4, llint z3, llint z4,
    llint lx, llint ly, llint lz,
    float hdx_2, float hdy_2, float hdz_2,
    float coef0,
    float coefx_1, float coefx_2, float coefx_3, float coefx_4,
    float coefy_1, float coefy_2, float coefy_3, float coefy_4,
    float coefz_1, float coefz_2, float coefz_3, float coefz_4,
    const float *__restrict__ u, float *__restrict__ v, const float *__restrict__ vp,
    const float *__restrict__ phi, const float *__restrict__ eta
) {
    __shared__ float s_u[N_THREADS_PER_PLANE_DIM_Y+2*N_RADIUS][N_THREADS_PER_PLANE_DIM_X+2*N_RADIUS];

    const llint j0 = y3 + blockIdx.y * blockDim.y;
    const llint k0 = z3 + blockIdx.x * blockDim.x;

    const llint je = min(j0 + N_THREADS_PER_PLANE_DIM_Y, y4);
    const llint ke = min(k0 + N_THREADS_PER_PLANE_DIM_X, z4);

    const llint j = j0 + threadIdx.y;
    const llint k = k0 + threadIdx.x;

    const llint sje = (j0+N_THREADS_PER_PLANE_DIM_Y<y4) ? N_THREADS_PER_PLANE_DIM_Y : ((y4-y3-1)%N_THREADS_PER_PLANE_DIM_Y+1);
    const llint ske = (k0+N_THREADS_PER_PLANE_DIM_X<z4) ? N_THREADS_PER_PLANE_DIM_X : ((z4-z3-1)%N_THREADS_PER_PLANE_DIM_X+1);

    const llint suj = threadIdx.y + N_RADIUS;
    const llint suk = threadIdx.x + N_RADIUS;

    float infront1, infront2, infront3, infront4; // variables for input “in front of” the current slice
    float behind1, behind2, behind3, behind4; // variables for input “behind” the current slice
    float current; // input value in the current slice

    behind3  = u[IDX3_l(x3-4,j,k)];
    behind2  = u[IDX3_l(x3-3,j,k)];
    behind1  = u[IDX3_l(x3-2,j,k)];
    current  = u[IDX3_l(x3-1,j,k)];
    infront1 = u[IDX3_l(x3+0,j,k)];
    infront2 = u[IDX3_l(x3+1,j,k)];
    infront3 = u[IDX3_l(x3+2,j,k)];
    infront4 = u[IDX3_l(x3+3,j,k)];

    for (llint i = x3; i < x4; i++) {
        // advance the slice (move the thread-front)
        behind4  = behind3;
        behind3  = behind2;
        behind2  = behind1;
        behind1  = current;
        current  = infront1;
        infront1 = infront2;
        infront2 = infront3;
        infront3 = infront4;
        infront4 = u[IDX3_l(i+N_RADIUS,j,k)];
        __syncthreads();

        if (threadIdx.y < N_RADIUS) {
            s_u[threadIdx.y][suk] = u[IDX3_l(i, j - N_RADIUS, k)];
            s_u[threadIdx.y+sje+N_RADIUS][suk] = u[IDX3_l(i, threadIdx.y+je, k)];
        }
        if (threadIdx.x < N_RADIUS) {
            s_u[suj][threadIdx.x] = u[IDX3_l(i,j,k - N_RADIUS)];
            s_u[suj][threadIdx.x+ske+N_RADIUS] = u[IDX3_l(i,j,threadIdx.x+ke)];
        }
        s_u[suj][suk] = u[IDX3_l(i,j,k)];
        __syncthreads();

        if (j < y4 && k < z4) {
            float lap = __fmaf_rn(coef0, current
                      , __fmaf_rn(coefx_1, __fadd_rn(infront1,behind1)
                      , __fmaf_rn(coefy_1, __fadd_rn(s_u[suj+1][suk],s_u[suj-1][suk])
                      , __fmaf_rn(coefz_1, __fadd_rn(s_u[suj][suk+1],s_u[suj][suk-1])
                      , __fmaf_rn(coefx_2, __fadd_rn(infront2,behind2)
                      , __fmaf_rn(coefy_2, __fadd_rn(s_u[suj+2][suk],s_u[suj-2][suk])
                      , __fmaf_rn(coefz_2, __fadd_rn(s_u[suj][suk+2],s_u[suj][suk-2])
                      , __fmaf_rn(coefx_3, __fadd_rn(infront3,behind3)
                      , __fmaf_rn(coefy_3, __fadd_rn(s_u[suj+3][suk],s_u[suj-3][suk])
                      , __fmaf_rn(coefz_3, __fadd_rn(s_u[suj][suk+3],s_u[suj][suk-3])
                      , __fmaf_rn(coefx_4, __fadd_rn(infront4,behind4)
                      , __fmaf_rn(coefy_4, __fadd_rn(s_u[suj+4][suk],s_u[suj-4][suk])
                      , __fmul_rn(coefz_4, __fadd_rn(s_u[suj][suk+4],s_u[suj][suk-4])
            )))))))))))));

            v[IDX3_l(i,j,k)] = __fmaf_rn(2.f, current,
                __fmaf_rn(vp[IDX3(i,j,k)], lap, -v[IDX3_l(i,j,k)])
            );
        }
    }
}

__global__ void target_pml_3d_kernel(
    llint nx, llint ny, llint nz,
    llint x3, llint x4, llint y3, llint y4, llint z3, llint z4,
    llint lx, llint ly, llint lz,
    float hdx_2, float hdy_2, float hdz_2,
    float coef0,
    float coefx_1, float coefx_2, float coefx_3, float coefx_4,
    float coefy_1, float coefy_2, float coefy_3, float coefy_4,
    float coefz_1, float coefz_2, float coefz_3, float coefz_4,
    const float *__restrict__ u, float *__restrict__ v, const float *__restrict__ vp,
    float *__restrict__ phi, const float *__restrict__ eta
) {
    __shared__ float s_u[N_THREADS_PER_PLANE_DIM_Y+2*N_RADIUS][N_THREADS_PER_PLANE_DIM_X+2*N_RADIUS];

    const llint j0 = y3 + blockIdx.y * blockDim.y;
    const llint k0 = z3 + blockIdx.x * blockDim.x;

    const llint je = min(j0 + N_THREADS_PER_PLANE_DIM_Y, y4);
    const llint ke = min(k0 + N_THREADS_PER_PLANE_DIM_X, z4);

    const llint j = j0 + threadIdx.y;
    const llint k = k0 + threadIdx.x;

    const llint sje = (j0+N_THREADS_PER_PLANE_DIM_Y<y4) ? N_THREADS_PER_PLANE_DIM_Y : ((y4-y3-1)%N_THREADS_PER_PLANE_DIM_Y+1);
    const llint ske = (k0+N_THREADS_PER_PLANE_DIM_X<z4) ? N_THREADS_PER_PLANE_DIM_X : ((z4-z3-1)%N_THREADS_PER_PLANE_DIM_X+1);

    const llint suj = threadIdx.y + N_RADIUS;
    const llint suk = threadIdx.x + N_RADIUS;

    float infront1, infront2, infront3, infront4; // variables for input “in front of” the current slice
    float behind1, behind2, behind3, behind4; // variables for input “behind” the current slice
    float current; // input value in the current slice

    behind3  = u[IDX3_l(x3-4,j,k)];
    behind2  = u[IDX3_l(x3-3,j,k)];
    behind1  = u[IDX3_l(x3-2,j,k)];
    current  = u[IDX3_l(x3-1,j,k)];
    infront1 = u[IDX3_l(x3+0,j,k)];
    infront2 = u[IDX3_l(x3+1,j,k)];
    infront3 = u[IDX3_l(x3+2,j,k)];
    infront4 = u[IDX3_l(x3+3,j,k)];

    for (llint i = x3; i < x4; i++) {
        // advance the slice (move the thread-front)
        behind4  = behind3;
        behind3  = behind2;
        behind2  = behind1;
        behind1  = current;
        current  = infront1;
        infront1 = infront2;
        infront2 = infront3;
        infront3 = infront4;
        infront4 = u[IDX3_l(i+N_RADIUS,j,k)];

        __syncthreads();

        if (threadIdx.y < N_RADIUS) {
            s_u[threadIdx.y][suk] = u[IDX3_l(i, j - N_RADIUS, k)];
            s_u[threadIdx.y+sje+N_RADIUS][suk] = u[IDX3_l(i, threadIdx.y+je, k)];
        }
        if (threadIdx.x < N_RADIUS) {
            s_u[suj][threadIdx.x] = u[IDX3_l(i,j,k - N_RADIUS)];
            s_u[suj][threadIdx.x+ske+N_RADIUS] = u[IDX3_l(i,j,threadIdx.x+ke)];
        }

        s_u[suj][suk] = u[IDX3_l(i,j,k)];

        __syncthreads();

        if (j < y4 && k < z4) {
            float lap = __fmaf_rn(coef0, current
                      , __fmaf_rn(coefx_1, __fadd_rn(infront1,behind1)
                      , __fmaf_rn(coefy_1, __fadd_rn(s_u[suj+1][suk],s_u[suj-1][suk])
                      , __fmaf_rn(coefz_1, __fadd_rn(s_u[suj][suk+1],s_u[suj][suk-1])
                      , __fmaf_rn(coefx_2, __fadd_rn(infront2,behind2)
                      , __fmaf_rn(coefy_2, __fadd_rn(s_u[suj+2][suk],s_u[suj-2][suk])
                      , __fmaf_rn(coefz_2, __fadd_rn(s_u[suj][suk+2],s_u[suj][suk-2])
                      , __fmaf_rn(coefx_3, __fadd_rn(infront3,behind3)
                      , __fmaf_rn(coefy_3, __fadd_rn(s_u[suj+3][suk],s_u[suj-3][suk])
                      , __fmaf_rn(coefz_3, __fadd_rn(s_u[suj][suk+3],s_u[suj][suk-3])
                      , __fmaf_rn(coefx_4, __fadd_rn(infront4,behind4)
                      , __fmaf_rn(coefy_4, __fadd_rn(s_u[suj+4][suk],s_u[suj-4][suk])
                      , __fmul_rn(coefz_4, __fadd_rn(s_u[suj][suk+4],s_u[suj][suk-4])
            )))))))))))));

            const float s_eta_c = eta[IDX3_eta1(i,j,k)];

            v[IDX3_l(i,j,k)] = __fdiv_rn(
                __fmaf_rn(
                    __fmaf_rn(2.f, s_eta_c,
                        __fsub_rn(2.f,
                            __fmul_rn(s_eta_c, s_eta_c)
                        )
                    ),
                    current,
                    __fmaf_rn(
                        vp[IDX3(i,j,k)],
                        __fadd_rn(lap, phi[IDX3(i,j,k)]),
                        -v[IDX3_l(i,j,k)]
                    )
                ),
                __fmaf_rn(2.f, s_eta_c, 1.f)
            );

            phi[IDX3(i,j,k)] = __fdiv_rn(
                    __fsub_rn(
                        phi[IDX3(i,j,k)],
                        __fmaf_rn(
                        __fmul_rn(
                            __fsub_rn(eta[IDX3_eta1(i+1,j,k)], eta[IDX3_eta1(i-1,j,k)]),
                            __fsub_rn(infront1,behind1)
                        ), hdx_2,
                        __fmaf_rn(
                        __fmul_rn(
                            __fsub_rn(eta[IDX3_eta1(i,j+1,k)], eta[IDX3_eta1(i,j-1,k)]),
                            __fsub_rn(s_u[suj+1][suk], s_u[suj-1][suk])
                        ), hdy_2,
                        __fmul_rn(
                            __fmul_rn(
                                __fsub_rn(eta[IDX3_eta1(i,j,k+1)], eta[IDX3_eta1(i,j,k-1)]),
                                __fsub_rn(s_u[suj][suk+1], s_u[suj][suk-1])
                            ),
                        hdz_2)
                        ))
                    )
                ,
                __fadd_rn(1.f, s_eta_c)
            );
        }
    }
}

__global__ void kernel_add_source_kernel(float *g_u, llint idx, float source) {
    g_u[idx] += source;
}

extern "C" void target(
    uint nsteps, double *time_kernel,
    llint nx, llint ny, llint nz,
    llint x1, llint x2, llint x3, llint x4, llint x5, llint x6,
    llint y1, llint y2, llint y3, llint y4, llint y5, llint y6,
    llint z1, llint z2, llint z3, llint z4, llint z5, llint z6,
    llint lx, llint ly, llint lz,
    llint sx, llint sy, llint sz,
    float hdx_2, float hdy_2, float hdz_2,
    const float *__restrict__ coefx, const float *__restrict__ coefy, const float *__restrict__ coefz,
    float *__restrict__ u, const float *__restrict__ v, const float *__restrict__ vp,
    const float *__restrict__ phi, const float *__restrict__ eta, const float *__restrict__ source
) {
    struct timespec start, end;

    const llint size_u = (nx + 2 * lx) * (ny + 2 * ly) * (nz + 2 * lz);
    const llint size_v = size_u;
    const llint size_phi = nx*ny*nz;
    const llint size_vp = size_phi;
    const llint size_eta = (nx+2)*(ny+2)*(nz+2);

    const llint size_u_ext = (nx + 2 * lx)
                           * ((((ny+N_THREADS_PER_PLANE_DIM_Y-1) / N_THREADS_PER_PLANE_DIM_Y + 1) * N_THREADS_PER_PLANE_DIM_Y) + 2 * ly)
                           * ((((nz+N_THREADS_PER_PLANE_DIM_X-1) / N_THREADS_PER_PLANE_DIM_X + 1) * N_THREADS_PER_PLANE_DIM_X) + 2 * lz);

    float *d_u, *d_v, *d_vp, *d_phi, *d_eta;
    hipMalloc(&d_u, sizeof(float) * size_u_ext);
    hipMalloc(&d_v, sizeof(float) * size_u_ext);
    hipMalloc(&d_vp, sizeof(float) * size_vp);
    hipMalloc(&d_phi, sizeof(float) * size_phi);
    hipMalloc(&d_eta, sizeof(float) * size_eta);

    hipMemcpy(d_u, u, sizeof(float) * size_u, hipMemcpyHostToDevice);
    hipMemcpy(d_v, v, sizeof(float) * size_v, hipMemcpyHostToDevice);
    hipMemcpy(d_vp, vp, sizeof(float) * size_vp, hipMemcpyHostToDevice);
    hipMemcpy(d_phi, phi, sizeof(float) * size_phi, hipMemcpyHostToDevice);
    hipMemcpy(d_eta, eta, sizeof(float) * size_eta, hipMemcpyHostToDevice);

    const llint xmin = 0; const llint xmax = nx;
    const llint ymin = 0; const llint ymax = ny;

    dim3 threadsPerBlock(N_THREADS_PER_PLANE_DIM_X, N_THREADS_PER_PLANE_DIM_Y, 1);

    int num_streams = 7;
    hipStream_t streams[num_streams];
    for (int i = 0; i < num_streams; i++) {
        hipStreamCreateWithFlags(&(streams[i]), hipStreamNonBlocking);
    }

    const uint npo = 100;
    for (uint istep = 1; istep <= nsteps; ++istep) {
        clock_gettime(CLOCK_REALTIME, &start);

        dim3 n_block_front(
            (z2-z1+N_THREADS_PER_PLANE_DIM_X-1) / N_THREADS_PER_PLANE_DIM_X,
            (ny+N_THREADS_PER_PLANE_DIM_Y-1) / N_THREADS_PER_PLANE_DIM_Y);
        target_pml_3d_kernel<<<n_block_front, threadsPerBlock, 0, streams[1]>>>(nx,ny,nz,
            xmin,xmax,ymin,ymax,z1,z2,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_top(
            (z4-z3+N_THREADS_PER_PLANE_DIM_X-1) / N_THREADS_PER_PLANE_DIM_X,
            (y2-y1+N_THREADS_PER_PLANE_DIM_Y-1) / N_THREADS_PER_PLANE_DIM_Y);
        target_pml_3d_kernel<<<n_block_top, threadsPerBlock, 0, streams[2]>>>(nx,ny,nz,
            xmin,xmax,y1,y2,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_left(
            (z4-z3+N_THREADS_PER_PLANE_DIM_X-1) / N_THREADS_PER_PLANE_DIM_X,
            (y4-y3+N_THREADS_PER_PLANE_DIM_Y-1) / N_THREADS_PER_PLANE_DIM_Y,
            1);
        target_pml_3d_kernel<<<n_block_left, threadsPerBlock, 0, streams[3]>>>(nx,ny,nz,
            x1,x2,y3,y4,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_center(
            (z4-z3+N_THREADS_PER_PLANE_DIM_X-1) / N_THREADS_PER_PLANE_DIM_X,
            (y4-y3+N_THREADS_PER_PLANE_DIM_Y-1) / N_THREADS_PER_PLANE_DIM_Y);
        target_inner_3d_kernel<<<n_block_center, threadsPerBlock, 0, streams[0]>>>(nx,ny,nz,
            x3,x4,y3,y4,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_right(
            (z4-z3+N_THREADS_PER_PLANE_DIM_X-1) / N_THREADS_PER_PLANE_DIM_X,
            (y4-y3+N_THREADS_PER_PLANE_DIM_Y-1) / N_THREADS_PER_PLANE_DIM_Y,
            1);
        target_pml_3d_kernel<<<n_block_right, threadsPerBlock, 0, streams[4]>>>(nx,ny,nz,
            x5,x6,y3,y4,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_bottom(
            (z4-z3+N_THREADS_PER_PLANE_DIM_X-1) / N_THREADS_PER_PLANE_DIM_X,
            (y6-y5+N_THREADS_PER_PLANE_DIM_Y-1) / N_THREADS_PER_PLANE_DIM_Y,
            1);
        target_pml_3d_kernel<<<n_block_bottom, threadsPerBlock, 0, streams[5]>>>(nx,ny,nz,
            xmin,xmax,y5,y6,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_back(
            (z6-z5+N_THREADS_PER_PLANE_DIM_X-1) / N_THREADS_PER_PLANE_DIM_X,
            (ny+N_THREADS_PER_PLANE_DIM_Y-1) / N_THREADS_PER_PLANE_DIM_Y,
            1);
        target_pml_3d_kernel<<<n_block_back, threadsPerBlock, 0, streams[6]>>>(nx,ny,nz,
            xmin,xmax,ymin,ymax,z5,z6,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        for (int i = 0; i < num_streams; i++) {
            hipStreamSynchronize(streams[i]);
        }

        kernel_add_source_kernel<<<1, 1>>>(d_v, IDX3_l(sx,sy,sz), source[istep]);
        clock_gettime(CLOCK_REALTIME, &end);
        *time_kernel += (end.tv_sec  - start.tv_sec) +
                        (double)(end.tv_nsec - start.tv_nsec) / 1.0e9;

        float *t = d_u;
        d_u = d_v;
        d_v = t;

        // Print out
        if (istep % npo == 0) {
            printf("time step %u / %u\n", istep, nsteps);
        }
    }


    for (int i = 0; i < num_streams; i++) {
        hipStreamDestroy(streams[i]);
    }


    hipMemcpy(u, d_u, sizeof(float) * size_u, hipMemcpyDeviceToHost);

    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_vp);
    hipFree(d_phi);
    hipFree(d_eta);
}
