#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <float.h>
#include <math.h>
#include <time.h>

#include "../../constants.h"

#define N_RADIUS 4
#define N_THREADS_PER_BLOCK_DIM 8
#define S_ETA_DIM  10
#define S_ETA_DIM2 100
#define S_ETA_SIZE 1000

__global__ void target_inner_3d_kernel(
    llint nx, llint ny, llint nz,
    llint x3, llint x4, llint y3, llint y4, llint z3, llint z4,
    llint lx, llint ly, llint lz,
    float hdx_2, float hdy_2, float hdz_2,
    float coef0,
    float coefx_1, float coefx_2, float coefx_3, float coefx_4,
    float coefy_1, float coefy_2, float coefy_3, float coefy_4,
    float coefz_1, float coefz_2, float coefz_3, float coefz_4,
    const float *__restrict__ u, float *__restrict__ v, const float *__restrict__ vp,
    const float *__restrict__ phi, const float *__restrict__ eta
) {
    const llint k0 = z3 + blockIdx.x * N_THREADS_PER_BLOCK_DIM;
    const llint j0 = y3 + blockIdx.y * N_THREADS_PER_BLOCK_DIM;
    const llint i0 = x3 + blockIdx.z * N_THREADS_PER_BLOCK_DIM;

    const llint i = i0 + threadIdx.z;
    const llint j = j0 + threadIdx.y;
    const llint k = k0 + threadIdx.x;

    if (i > x4-1 || j > y4-1 || k > z4-1) { return; }

    float lap = __fmaf_rn(coef0, u[IDX3_l(i,j,k)]
              , __fmaf_rn(coefx_1, __fadd_rn(u[IDX3_l(i+1,j,k)],u[IDX3_l(i-1,j,k)])
              , __fmaf_rn(coefy_1, __fadd_rn(u[IDX3_l(i,j+1,k)],u[IDX3_l(i,j-1,k)])
              , __fmaf_rn(coefz_1, __fadd_rn(u[IDX3_l(i,j,k+1)],u[IDX3_l(i,j,k-1)])
              , __fmaf_rn(coefx_2, __fadd_rn(u[IDX3_l(i+2,j,k)],u[IDX3_l(i-2,j,k)])
              , __fmaf_rn(coefy_2, __fadd_rn(u[IDX3_l(i,j+2,k)],u[IDX3_l(i,j-2,k)])
              , __fmaf_rn(coefz_2, __fadd_rn(u[IDX3_l(i,j,k+2)],u[IDX3_l(i,j,k-2)])
              , __fmaf_rn(coefx_3, __fadd_rn(u[IDX3_l(i+3,j,k)],u[IDX3_l(i-3,j,k)])
              , __fmaf_rn(coefy_3, __fadd_rn(u[IDX3_l(i,j+3,k)],u[IDX3_l(i,j-3,k)])
              , __fmaf_rn(coefz_3, __fadd_rn(u[IDX3_l(i,j,k+3)],u[IDX3_l(i,j,k-3)])
              , __fmaf_rn(coefx_4, __fadd_rn(u[IDX3_l(i+4,j,k)],u[IDX3_l(i-4,j,k)])
              , __fmaf_rn(coefy_4, __fadd_rn(u[IDX3_l(i,j+4,k)],u[IDX3_l(i,j-4,k)])
              , __fmul_rn(coefz_4, __fadd_rn(u[IDX3_l(i,j,k+4)],u[IDX3_l(i,j,k-4)])
    )))))))))))));

    v[IDX3_l(i,j,k)] = __fmaf_rn(2.f, u[IDX3_l(i,j,k)],
        __fmaf_rn(vp[IDX3(i,j,k)], lap, -v[IDX3_l(i,j,k)])
    );
}

__global__ void target_pml_3d_kernel(
    llint nx, llint ny, llint nz,
    llint x3, llint x4, llint y3, llint y4, llint z3, llint z4,
    llint lx, llint ly, llint lz,
    float hdx_2, float hdy_2, float hdz_2,
    float coef0,
    float coefx_1, float coefx_2, float coefx_3, float coefx_4,
    float coefy_1, float coefy_2, float coefy_3, float coefy_4,
    float coefz_1, float coefz_2, float coefz_3, float coefz_4,
    const float *__restrict__ u, float *__restrict__ v, const float *__restrict__ vp,
    float *__restrict__ phi, const float *__restrict__ eta
) {
    __shared__ float s_eta[S_ETA_SIZE];

    const llint k0 = z3 + blockIdx.x * N_THREADS_PER_BLOCK_DIM;
    const llint j0 = y3 + blockIdx.y * N_THREADS_PER_BLOCK_DIM;
    const llint i0 = x3 + blockIdx.z * N_THREADS_PER_BLOCK_DIM;

    const llint i = i0 + threadIdx.z;
    const llint j = j0 + threadIdx.y;
    const llint k = k0 + threadIdx.x;

    // s_eta[threadIdx.z][threadIdx.y][threadIdx.x] = eta[IDX3_eta1(i-1,j-1,k-1)];

    const llint setai = threadIdx.z + 1;
    const llint setaj = threadIdx.y + 1;
    const llint setak = threadIdx.x + 1;

    s_eta[setai*S_ETA_DIM2 + setaj*S_ETA_DIM + setak] = eta[IDX3_eta1(i,j,k)];

    // if (threadIdx.z < 6) {
    //     int z = threadIdx.z & 1;
    //     int q = (threadIdx.z & 2) == 2;
    //     int p = (threadIdx.z & 4) == 4;
    //     int z9 = z*9;
    //     int zz = (z * (N_THREADS_PER_BLOCK_DIM+1)) - 1;
    //     int npnq = (1-p)*(1-q);
    //     int porq = p|q;
    //     /*
    //         p | q | npnq | porq | 1-porq | (1-p)q | p(1-q)
    //         0   0    1      0       1        0        0
    //         0   1    0      1       0        1        0
    //         1   0    0      1       0        0        1
    //      */
    //     s_eta[
    //         ((npnq*z9) + (1-npnq)*setaj) * S_ETA_DIM2 +
    //         ((1-porq)*setaj + (1-p)*q*z9 + p*(1-q)*setak) * S_ETA_DIM +
    //         ((1-p)*setak + p*z9)
    //     ] = eta[IDX3_eta1(i0 + ( (llint)npnq*zz + (llint)(1-npnq)*threadIdx.y ), j0 + ( (llint)(1-porq)*threadIdx.y + (llint)(1-p)*(llint)(q)*zz + (llint)p*(llint)(1-q)*threadIdx.x ), k0 + ( (llint)(1-p)*threadIdx.x + (llint)p*zz ) )];
    // }

    if (threadIdx.z < 2) s_eta[threadIdx.z*9*S_ETA_DIM2 + setaj*S_ETA_DIM + setak] = eta[IDX3_eta1(i0 + (threadIdx.z * (N_THREADS_PER_BLOCK_DIM+1)) - 1, j, k)];
    if (threadIdx.y < 2) s_eta[setai*S_ETA_DIM2 + threadIdx.y*9*S_ETA_DIM + setak] = eta[IDX3_eta1(i, j0 + (threadIdx.y * (N_THREADS_PER_BLOCK_DIM+1)) - 1, k)];
    if (threadIdx.x < 2) s_eta[setai*S_ETA_DIM2 + setaj*S_ETA_DIM + threadIdx.x*9] = eta[IDX3_eta1(i, j, k0 + (threadIdx.x * (N_THREADS_PER_BLOCK_DIM+1)) - 1)];

    // if (threadIdx.x >= N_THREADS_PER_BLOCK_DIM || threadIdx.y >= N_THREADS_PER_BLOCK_DIM || threadIdx.z >= N_THREADS_PER_BLOCK_DIM) { return; }

    if (i > x4-1 || j > y4-1 || k > z4-1) { return; }

    // const llint setai = threadIdx.z + 1;
    // const llint setaj = threadIdx.y + 1;
    // const llint setak = threadIdx.x + 1;

    __syncthreads();

    float lap = __fmaf_rn(coef0, u[IDX3_l(i,j,k)]
              , __fmaf_rn(coefx_1, __fadd_rn(u[IDX3_l(i+1,j,k)],u[IDX3_l(i-1,j,k)])
              , __fmaf_rn(coefy_1, __fadd_rn(u[IDX3_l(i,j+1,k)],u[IDX3_l(i,j-1,k)])
              , __fmaf_rn(coefz_1, __fadd_rn(u[IDX3_l(i,j,k+1)],u[IDX3_l(i,j,k-1)])
              , __fmaf_rn(coefx_2, __fadd_rn(u[IDX3_l(i+2,j,k)],u[IDX3_l(i-2,j,k)])
              , __fmaf_rn(coefy_2, __fadd_rn(u[IDX3_l(i,j+2,k)],u[IDX3_l(i,j-2,k)])
              , __fmaf_rn(coefz_2, __fadd_rn(u[IDX3_l(i,j,k+2)],u[IDX3_l(i,j,k-2)])
              , __fmaf_rn(coefx_3, __fadd_rn(u[IDX3_l(i+3,j,k)],u[IDX3_l(i-3,j,k)])
              , __fmaf_rn(coefy_3, __fadd_rn(u[IDX3_l(i,j+3,k)],u[IDX3_l(i,j-3,k)])
              , __fmaf_rn(coefz_3, __fadd_rn(u[IDX3_l(i,j,k+3)],u[IDX3_l(i,j,k-3)])
              , __fmaf_rn(coefx_4, __fadd_rn(u[IDX3_l(i+4,j,k)],u[IDX3_l(i-4,j,k)])
              , __fmaf_rn(coefy_4, __fadd_rn(u[IDX3_l(i,j+4,k)],u[IDX3_l(i,j-4,k)])
              , __fmul_rn(coefz_4, __fadd_rn(u[IDX3_l(i,j,k+4)],u[IDX3_l(i,j,k-4)])
    )))))))))))));

    const float s_eta_c = s_eta[setai*S_ETA_DIM2 + setaj*S_ETA_DIM + setak];

    v[IDX3_l(i,j,k)] = __fdiv_rn(
        __fmaf_rn(
            __fmaf_rn(2.f, s_eta_c,
                __fsub_rn(2.f,
                    __fmul_rn(s_eta_c, s_eta_c)
                )
            ),
            u[IDX3_l(i,j,k)],
            __fmaf_rn(
                vp[IDX3(i,j,k)],
                __fadd_rn(lap, phi[IDX3(i,j,k)]),
                -v[IDX3_l(i,j,k)]
            )
        ),
        __fmaf_rn(2.f, s_eta_c, 1.f)
    );

    phi[IDX3(i,j,k)] = __fdiv_rn(
            __fsub_rn(
                phi[IDX3(i,j,k)],
                __fmaf_rn(
                __fmul_rn(
                    __fsub_rn(s_eta[(setai+1)*S_ETA_DIM2 + setaj*S_ETA_DIM + setak], s_eta[(setai-1)*S_ETA_DIM2 + setaj*S_ETA_DIM + setak]),
                    __fsub_rn(u[IDX3_l(i+1,j,k)], u[IDX3_l(i-1,j,k)])
                ), hdx_2,
                __fmaf_rn(
                __fmul_rn(
                    __fsub_rn(s_eta[setai*S_ETA_DIM2 + (setaj+1)*S_ETA_DIM + setak], s_eta[setai*S_ETA_DIM2 + (setaj-1)*S_ETA_DIM + setak]),
                    __fsub_rn(u[IDX3_l(i,j+1,k)], u[IDX3_l(i,j-1,k)])
                ), hdy_2,
                __fmul_rn(
                    __fmul_rn(
                        __fsub_rn(s_eta[setai*S_ETA_DIM2 + setaj*S_ETA_DIM + setak+1], s_eta[setai*S_ETA_DIM2 + setaj*S_ETA_DIM + setak-1]),
                        __fsub_rn(u[IDX3_l(i,j,k+1)], u[IDX3_l(i,j,k-1)])
                    ),
                hdz_2)
                ))
            )
        ,
        __fadd_rn(1.f, s_eta_c)
    );
}

__global__ void kernel_add_source_kernel(float *g_u, llint idx, float source) {
    g_u[idx] += source;
}

extern "C" void target(
    uint nsteps, double *time_kernel,
    llint nx, llint ny, llint nz,
    llint x1, llint x2, llint x3, llint x4, llint x5, llint x6,
    llint y1, llint y2, llint y3, llint y4, llint y5, llint y6,
    llint z1, llint z2, llint z3, llint z4, llint z5, llint z6,
    llint lx, llint ly, llint lz,
    llint sx, llint sy, llint sz,
    float hdx_2, float hdy_2, float hdz_2,
    const float *__restrict__ coefx, const float *__restrict__ coefy, const float *__restrict__ coefz,
    float *__restrict__ u, const float *__restrict__ v, const float *__restrict__ vp,
    const float *__restrict__ phi, const float *__restrict__ eta, const float *__restrict__ source
) {
    struct timespec start, end;

    const llint size_u = (nx + 2 * lx) * (ny + 2 * ly) * (nz + 2 * lz);
    const llint size_v = size_u;
    const llint size_phi = nx*ny*nz;
    const llint size_vp = size_phi;
    const llint size_eta = (nx+2)*(ny+2)*(nz+2);
    const llint size_eta_ext = ((((nx+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM + 1) * N_THREADS_PER_BLOCK_DIM)+2)
                             * ((((ny+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM + 1) * N_THREADS_PER_BLOCK_DIM)+2)
                             * ((((nz+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM + 1) * N_THREADS_PER_BLOCK_DIM)+2);

    float *d_u, *d_v, *d_vp, *d_phi, *d_eta;
    hipMalloc(&d_u, sizeof(float) * size_u);
    hipMalloc(&d_v, sizeof(float) * size_u);
    hipMalloc(&d_vp, sizeof(float) * size_vp);
    hipMalloc(&d_phi, sizeof(float) * size_phi);
    hipMalloc(&d_eta, sizeof(float) * size_eta_ext);

    hipMemcpy(d_u, u, sizeof(float) * size_u, hipMemcpyHostToDevice);
    hipMemcpy(d_v, v, sizeof(float) * size_v, hipMemcpyHostToDevice);
    hipMemcpy(d_vp, vp, sizeof(float) * size_vp, hipMemcpyHostToDevice);
    hipMemcpy(d_phi, phi, sizeof(float) * size_phi, hipMemcpyHostToDevice);
    hipMemcpy(d_eta, eta, sizeof(float) * size_eta, hipMemcpyHostToDevice);

    const llint xmin = 0; const llint xmax = nx;
    const llint ymin = 0; const llint ymax = ny;

    dim3 threadsPerBlock(N_THREADS_PER_BLOCK_DIM, N_THREADS_PER_BLOCK_DIM, N_THREADS_PER_BLOCK_DIM);
    // dim3 threadsPerBlock(N_THREADS_PER_BLOCK_DIM+2, N_THREADS_PER_BLOCK_DIM+2, N_THREADS_PER_BLOCK_DIM+2);
    dim3 threadsPerBlock_inner(N_THREADS_PER_BLOCK_DIM, N_THREADS_PER_BLOCK_DIM, N_THREADS_PER_BLOCK_DIM);

    int num_streams = 7;
    hipStream_t streams[num_streams];
    for (int i = 0; i < num_streams; i++) {
        hipStreamCreate(&(streams[i]));
    }

    const uint npo = 100;
    for (uint istep = 1; istep <= nsteps; ++istep) {
        clock_gettime(CLOCK_REALTIME, &start);

        dim3 n_block_front(
            (z2-z1+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (ny+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (nx+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM);
        target_pml_3d_kernel<<<n_block_front, threadsPerBlock, 0, streams[1]>>>(nx,ny,nz,
            xmin,xmax,ymin,ymax,z1,z2,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_top(
            (z4-z3+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (y2-y1+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (nx+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM);
        target_pml_3d_kernel<<<n_block_top, threadsPerBlock, 0, streams[2]>>>(nx,ny,nz,
            xmin,xmax,y1,y2,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_left(
            (z4-z3+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (y4-y3+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (x2-x1+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM);
        target_pml_3d_kernel<<<n_block_left, threadsPerBlock, 0, streams[3]>>>(nx,ny,nz,
            x1,x2,y3,y4,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_center(
            (z4-z3+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (y4-y3+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (x4-x3+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM);
        target_inner_3d_kernel<<<n_block_center, threadsPerBlock_inner, 0, streams[0]>>>(nx,ny,nz,
            x3,x4,y3,y4,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_right(
            (z4-z3+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (y4-y3+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (x6-x5+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM);
        target_pml_3d_kernel<<<n_block_right, threadsPerBlock, 0, streams[4]>>>(nx,ny,nz,
            x5,x6,y3,y4,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_bottom(
            (z4-z3+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (y6-y5+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (nx+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM);
        target_pml_3d_kernel<<<n_block_bottom, threadsPerBlock, 0, streams[5]>>>(nx,ny,nz,
            xmin,xmax,y5,y6,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_back(
            (z6-z5+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (ny+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (nx+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM);
        target_pml_3d_kernel<<<n_block_back, threadsPerBlock, 0, streams[6]>>>(nx,ny,nz,
            xmin,xmax,ymin,ymax,z5,z6,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        for (int i = 0; i < num_streams; i++) {
            hipStreamSynchronize(streams[i]);
        }

        kernel_add_source_kernel<<<1, 1>>>(d_v, IDX3_l(sx,sy,sz), source[istep]);
        clock_gettime(CLOCK_REALTIME, &end);
        *time_kernel += (end.tv_sec  - start.tv_sec) +
                        (double)(end.tv_nsec - start.tv_nsec) / 1.0e9;

        float *t = d_u;
        d_u = d_v;
        d_v = t;

        // Print out
        if (istep % npo == 0) {
            printf("time step %u / %u\n", istep, nsteps);
        }
    }


    for (int i = 0; i < num_streams; i++) {
        hipStreamDestroy(streams[i]);
    }


    hipMemcpy(u, d_u, sizeof(float) * size_u, hipMemcpyDeviceToHost);

    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_vp);
    hipFree(d_phi);
    hipFree(d_eta);
}

