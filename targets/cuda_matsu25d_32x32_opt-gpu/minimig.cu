#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <float.h>
#include <math.h>
#include <time.h>

#include "../../constants.h"

#define N_RADIUS 4
#define N_THREADS_PER_PLANE_DIM_X 32
#define N_THREADS_PER_PLANE_DIM_Y 32

#define Z(val) ((val > 8) ? (val - 9) : val)

#define INNER_STENCIL(R0, R1, R2, R3, R4, R5, R6, R7, R8) i++; if (i >= x4) { return; } \
    zrs##R8 = u[IDX3_l(i+N_RADIUS,j,k)]; \
    __syncthreads(); \
    if (threadIdx.y < 2 * N_RADIUS) { \
        s_u[threadIdx.y + (threadIdx.y/N_RADIUS)*N_THREADS_PER_PLANE_DIM_Y][suk] = \
            u[IDX3_l(i,j0+threadIdx.y+(threadIdx.y/N_RADIUS)*N_THREADS_PER_PLANE_DIM_Y-N_RADIUS,k)]; \
    } \
    if (threadIdx.x < 2 * N_RADIUS) { \
        s_u[suj][threadIdx.x + (threadIdx.x/N_RADIUS)*N_THREADS_PER_PLANE_DIM_X] = \
            u[IDX3_l(i,j,k0+threadIdx.x+(threadIdx.x/N_RADIUS)*N_THREADS_PER_PLANE_DIM_X-N_RADIUS)]; \
    } \
    s_u[suj][suk] = u[IDX3_l(i,j,k)]; \
    __syncthreads(); \
    if (j < y4 && k < z4) { \
        float lap = __fmaf_rn(coef0, zrs##R4 \
                    , __fmaf_rn(coefx_1, __fadd_rn(zrs##R5,zrs##R3) \
                    , __fmaf_rn(coefy_1, __fadd_rn(s_u[suj+1][suk],s_u[suj-1][suk]) \
                    , __fmaf_rn(coefz_1, __fadd_rn(s_u[suj][suk+1],s_u[suj][suk-1]) \
                    , __fmaf_rn(coefx_2, __fadd_rn(zrs##R6,zrs##R2) \
                    , __fmaf_rn(coefy_2, __fadd_rn(s_u[suj+2][suk],s_u[suj-2][suk]) \
                    , __fmaf_rn(coefz_2, __fadd_rn(s_u[suj][suk+2],s_u[suj][suk-2]) \
                    , __fmaf_rn(coefx_3, __fadd_rn(zrs##R7,zrs##R1) \
                    , __fmaf_rn(coefy_3, __fadd_rn(s_u[suj+3][suk],s_u[suj-3][suk]) \
                    , __fmaf_rn(coefz_3, __fadd_rn(s_u[suj][suk+3],s_u[suj][suk-3]) \
                    , __fmaf_rn(coefx_4, __fadd_rn(zrs##R8,zrs##R0) \
                    , __fmaf_rn(coefy_4, __fadd_rn(s_u[suj+4][suk],s_u[suj-4][suk]) \
                    , __fmul_rn(coefz_4, __fadd_rn(s_u[suj][suk+4],s_u[suj][suk-4]) \
        ))))))))))))); \
        v[IDX3_l(i,j,k)] = __fmaf_rn(2.f, zrs##R4, \
            __fmaf_rn(vp[IDX3(i,j,k)], lap, -v[IDX3_l(i,j,k)]) \
        ); \
    }

#define PML_STENCIL(R0, R1, R2, R3, R4, R5, R6, R7, R8) i++; if (i >= x4) { return; } \
    zrs##R8 = u[IDX3_l(i+N_RADIUS,j,k)]; \
    __syncthreads(); \
    if (threadIdx.y < 2 * N_RADIUS) { \
        s_u[threadIdx.y + (threadIdx.y/N_RADIUS)*N_THREADS_PER_PLANE_DIM_Y][suk] = \
            u[IDX3_l(i,j0+threadIdx.y+(threadIdx.y/N_RADIUS)*N_THREADS_PER_PLANE_DIM_Y-N_RADIUS,k)]; \
    } \
    if (threadIdx.x < 2 * N_RADIUS) { \
        s_u[suj][threadIdx.x + (threadIdx.x/N_RADIUS)*N_THREADS_PER_PLANE_DIM_X] = \
            u[IDX3_l(i,j,k0+threadIdx.x+(threadIdx.x/N_RADIUS)*N_THREADS_PER_PLANE_DIM_X-N_RADIUS)]; \
    } \
    s_u[suj][suk] = u[IDX3_l(i,j,k)]; \
    __syncthreads(); \
    if (j < y4 && k < z4) { \
        float lap = __fmaf_rn(coef0, zrs##R4 \
                    , __fmaf_rn(coefx_1, __fadd_rn(zrs##R5,zrs##R3) \
                    , __fmaf_rn(coefy_1, __fadd_rn(s_u[suj+1][suk],s_u[suj-1][suk]) \
                    , __fmaf_rn(coefz_1, __fadd_rn(s_u[suj][suk+1],s_u[suj][suk-1]) \
                    , __fmaf_rn(coefx_2, __fadd_rn(zrs##R6,zrs##R2) \
                    , __fmaf_rn(coefy_2, __fadd_rn(s_u[suj+2][suk],s_u[suj-2][suk]) \
                    , __fmaf_rn(coefz_2, __fadd_rn(s_u[suj][suk+2],s_u[suj][suk-2]) \
                    , __fmaf_rn(coefx_3, __fadd_rn(zrs##R7,zrs##R1) \
                    , __fmaf_rn(coefy_3, __fadd_rn(s_u[suj+3][suk],s_u[suj-3][suk]) \
                    , __fmaf_rn(coefz_3, __fadd_rn(s_u[suj][suk+3],s_u[suj][suk-3]) \
                    , __fmaf_rn(coefx_4, __fadd_rn(zrs##R8,zrs##R0) \
                    , __fmaf_rn(coefy_4, __fadd_rn(s_u[suj+4][suk],s_u[suj-4][suk]) \
                    , __fmul_rn(coefz_4, __fadd_rn(s_u[suj][suk+4],s_u[suj][suk-4]) \
        ))))))))))))); \
        const float s_eta_c = eta[IDX3_eta1(i,j,k)]; \
        v[IDX3_l(i,j,k)] = __fdiv_rn( \
            __fmaf_rn( \
                __fmaf_rn(2.f, s_eta_c, \
                    __fsub_rn(2.f, \
                        __fmul_rn(s_eta_c, s_eta_c) \
                    ) \
                ), \
                zrs##R4, \
                __fmaf_rn( \
                    vp[IDX3(i,j,k)], \
                    __fadd_rn(lap, phi[IDX3(i,j,k)]), \
                    -v[IDX3_l(i,j,k)] \
                ) \
            ), \
            __fmaf_rn(2.f, s_eta_c, 1.f) \
        ); \
        phi[IDX3(i,j,k)] = __fdiv_rn( \
                __fsub_rn( \
                    phi[IDX3(i,j,k)], \
                    __fmaf_rn( \
                    __fmul_rn( \
                        __fsub_rn(eta[IDX3_eta1(i+1,j,k)], eta[IDX3_eta1(i-1,j,k)]), \
                        __fsub_rn(zrs##R5,zrs##R3) \
                    ), hdx_2, \
                    __fmaf_rn( \
                    __fmul_rn( \
                        __fsub_rn(eta[IDX3_eta1(i,j+1,k)], eta[IDX3_eta1(i,j-1,k)]), \
                        __fsub_rn(s_u[suj+1][suk], s_u[suj-1][suk]) \
                    ), hdy_2, \
                    __fmul_rn( \
                        __fmul_rn( \
                            __fsub_rn(eta[IDX3_eta1(i,j,k+1)], eta[IDX3_eta1(i,j,k-1)]), \
                            __fsub_rn(s_u[suj][suk+1], s_u[suj][suk-1]) \
                        ), \
                    hdz_2) \
                    )) \
                ) \
            , \
            __fadd_rn(1.f, s_eta_c) \
        ); \
    }

__global__ void target_inner_3d_kernel(
    llint nx, llint ny, llint nz,
    llint x3, llint x4, llint y3, llint y4, llint z3, llint z4,
    llint lx, llint ly, llint lz,
    float hdx_2, float hdy_2, float hdz_2,
    float coef0,
    float coefx_1, float coefx_2, float coefx_3, float coefx_4,
    float coefy_1, float coefy_2, float coefy_3, float coefy_4,
    float coefz_1, float coefz_2, float coefz_3, float coefz_4,
    const float *__restrict__ u, float *__restrict__ v, const float *__restrict__ vp,
    const float *__restrict__ phi, const float *__restrict__ eta
) {
    __shared__ float s_u[N_THREADS_PER_PLANE_DIM_Y+2*N_RADIUS][N_THREADS_PER_PLANE_DIM_X+2*N_RADIUS];

    const llint j0 = y3 + blockIdx.y * blockDim.y;
    const llint k0 = z3 + blockIdx.x * blockDim.x;

    const llint j = j0 + threadIdx.y;
    const llint k = k0 + threadIdx.x;

    const llint suj = threadIdx.y + N_RADIUS;
    const llint suk = threadIdx.x + N_RADIUS;

    float zrs0, zrs1, zrs2, zrs3, zrs4, zrs5, zrs6, zrs7, zrs8;

    // Preparation
    zrs0 = u[IDX3_l(x3-4,j,k)];
    zrs1 = u[IDX3_l(x3-3,j,k)];
    zrs2 = u[IDX3_l(x3-2,j,k)];
    zrs3 = u[IDX3_l(x3-1,j,k)];
    zrs4 = u[IDX3_l(x3+0,j,k)];
    zrs5 = u[IDX3_l(x3+1,j,k)];
    zrs6 = u[IDX3_l(x3+2,j,k)];
    zrs7 = u[IDX3_l(x3+3,j,k)];

    llint i = x3-1;
    while (true) {
        INNER_STENCIL(0, 1, 2, 3, 4, 5, 6, 7, 8);
        INNER_STENCIL(1, 2, 3, 4, 5, 6, 7, 8, 0);
        INNER_STENCIL(2, 3, 4, 5, 6, 7, 8, 0, 1);
        INNER_STENCIL(3, 4, 5, 6, 7, 8, 0, 1, 2);
        INNER_STENCIL(4, 5, 6, 7, 8, 0, 1, 2, 3);
        INNER_STENCIL(5, 6, 7, 8, 0, 1, 2, 3, 4);
        INNER_STENCIL(6, 7, 8, 0, 1, 2, 3, 4, 5);
        INNER_STENCIL(7, 8, 0, 1, 2, 3, 4, 5, 6);
        INNER_STENCIL(8, 0, 1, 2, 3, 4, 5, 6, 7);
    }
}

__global__ void target_pml_3d_kernel(
    llint nx, llint ny, llint nz,
    llint x3, llint x4, llint y3, llint y4, llint z3, llint z4,
    llint lx, llint ly, llint lz,
    float hdx_2, float hdy_2, float hdz_2,
    float coef0,
    float coefx_1, float coefx_2, float coefx_3, float coefx_4,
    float coefy_1, float coefy_2, float coefy_3, float coefy_4,
    float coefz_1, float coefz_2, float coefz_3, float coefz_4,
    const float *__restrict__ u, float *__restrict__ v, const float *__restrict__ vp,
    float *__restrict__ phi, const float *__restrict__ eta
) {
    __shared__ float s_u[N_THREADS_PER_PLANE_DIM_Y+2*N_RADIUS][N_THREADS_PER_PLANE_DIM_X+2*N_RADIUS];

    const llint j0 = y3 + blockIdx.y * blockDim.y;
    const llint k0 = z3 + blockIdx.x * blockDim.x;

    const llint j = j0 + threadIdx.y;
    const llint k = k0 + threadIdx.x;

    const llint suj = threadIdx.y + N_RADIUS;
    const llint suk = threadIdx.x + N_RADIUS;

    float zrs0, zrs1, zrs2, zrs3, zrs4, zrs5, zrs6, zrs7, zrs8;

    // Preparation
    zrs0 = u[IDX3_l(x3-4,j,k)];
    zrs1 = u[IDX3_l(x3-3,j,k)];
    zrs2 = u[IDX3_l(x3-2,j,k)];
    zrs3 = u[IDX3_l(x3-1,j,k)];
    zrs4 = u[IDX3_l(x3+0,j,k)];
    zrs5 = u[IDX3_l(x3+1,j,k)];
    zrs6 = u[IDX3_l(x3+2,j,k)];
    zrs7 = u[IDX3_l(x3+3,j,k)];

    llint i = x3-1;
    while (true) {
        PML_STENCIL(0, 1, 2, 3, 4, 5, 6, 7, 8);
        PML_STENCIL(1, 2, 3, 4, 5, 6, 7, 8, 0);
        PML_STENCIL(2, 3, 4, 5, 6, 7, 8, 0, 1);
        PML_STENCIL(3, 4, 5, 6, 7, 8, 0, 1, 2);
        PML_STENCIL(4, 5, 6, 7, 8, 0, 1, 2, 3);
        PML_STENCIL(5, 6, 7, 8, 0, 1, 2, 3, 4);
        PML_STENCIL(6, 7, 8, 0, 1, 2, 3, 4, 5);
        PML_STENCIL(7, 8, 0, 1, 2, 3, 4, 5, 6);
        PML_STENCIL(8, 0, 1, 2, 3, 4, 5, 6, 7);
    }
}

__global__ void kernel_add_source_kernel(float *g_u, llint idx, float source) {
    g_u[idx] += source;
}

extern "C" void target(
    uint nsteps, double *time_kernel,
    llint nx, llint ny, llint nz,
    llint x1, llint x2, llint x3, llint x4, llint x5, llint x6,
    llint y1, llint y2, llint y3, llint y4, llint y5, llint y6,
    llint z1, llint z2, llint z3, llint z4, llint z5, llint z6,
    llint lx, llint ly, llint lz,
    llint sx, llint sy, llint sz,
    float hdx_2, float hdy_2, float hdz_2,
    const float *__restrict__ coefx, const float *__restrict__ coefy, const float *__restrict__ coefz,
    float *__restrict__ u, const float *__restrict__ v, const float *__restrict__ vp,
    const float *__restrict__ phi, const float *__restrict__ eta, const float *__restrict__ source
) {
    struct timespec start, end;

    const llint size_u = (nx + 2 * lx) * (ny + 2 * ly) * (nz + 2 * lz);
    const llint size_v = size_u;
    const llint size_phi = nx*ny*nz;
    const llint size_vp = size_phi;
    const llint size_eta = (nx+2)*(ny+2)*(nz+2);

    const llint size_u_ext = (nx + 2 * lx)
                           * ((((ny+N_THREADS_PER_PLANE_DIM_Y-1) / N_THREADS_PER_PLANE_DIM_Y + 1) * N_THREADS_PER_PLANE_DIM_Y) + 2 * ly)
                           * ((((nz+N_THREADS_PER_PLANE_DIM_X-1) / N_THREADS_PER_PLANE_DIM_X + 1) * N_THREADS_PER_PLANE_DIM_X) + 2 * lz);

    float *d_u, *d_v, *d_vp, *d_phi, *d_eta;
    hipMalloc(&d_u, sizeof(float) * size_u_ext);
    hipMalloc(&d_v, sizeof(float) * size_u_ext);
    hipMalloc(&d_vp, sizeof(float) * size_vp);
    hipMalloc(&d_phi, sizeof(float) * size_phi);
    hipMalloc(&d_eta, sizeof(float) * size_eta);

    hipMemcpy(d_u, u, sizeof(float) * size_u, hipMemcpyHostToDevice);
    hipMemcpy(d_v, v, sizeof(float) * size_v, hipMemcpyHostToDevice);
    hipMemcpy(d_vp, vp, sizeof(float) * size_vp, hipMemcpyHostToDevice);
    hipMemcpy(d_phi, phi, sizeof(float) * size_phi, hipMemcpyHostToDevice);
    hipMemcpy(d_eta, eta, sizeof(float) * size_eta, hipMemcpyHostToDevice);

    const llint xmin = 0; const llint xmax = nx;
    const llint ymin = 0; const llint ymax = ny;

    dim3 threadsPerBlock(N_THREADS_PER_PLANE_DIM_X, N_THREADS_PER_PLANE_DIM_Y, 1);

    int num_streams = 7;
    hipStream_t streams[num_streams];
    for (int i = 0; i < num_streams; i++) {
        hipStreamCreateWithFlags(&(streams[i]), hipStreamNonBlocking);
    }

    const uint npo = 100;
    for (uint istep = 1; istep <= nsteps; ++istep) {
        clock_gettime(CLOCK_REALTIME, &start);

        dim3 n_block_front(
            (z2-z1+N_THREADS_PER_PLANE_DIM_X-1) / N_THREADS_PER_PLANE_DIM_X,
            (ny+N_THREADS_PER_PLANE_DIM_Y-1) / N_THREADS_PER_PLANE_DIM_Y);
        target_pml_3d_kernel<<<n_block_front, threadsPerBlock, 0, streams[1]>>>(nx,ny,nz,
            xmin,xmax,ymin,ymax,z1,z2,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_top(
            (z4-z3+N_THREADS_PER_PLANE_DIM_X-1) / N_THREADS_PER_PLANE_DIM_X,
            (y2-y1+N_THREADS_PER_PLANE_DIM_Y-1) / N_THREADS_PER_PLANE_DIM_Y);
        target_pml_3d_kernel<<<n_block_top, threadsPerBlock, 0, streams[2]>>>(nx,ny,nz,
            xmin,xmax,y1,y2,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_left(
            (z4-z3+N_THREADS_PER_PLANE_DIM_X-1) / N_THREADS_PER_PLANE_DIM_X,
            (y4-y3+N_THREADS_PER_PLANE_DIM_Y-1) / N_THREADS_PER_PLANE_DIM_Y,
            1);
        target_pml_3d_kernel<<<n_block_left, threadsPerBlock, 0, streams[3]>>>(nx,ny,nz,
            x1,x2,y3,y4,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_center(
            (z4-z3+N_THREADS_PER_PLANE_DIM_X-1) / N_THREADS_PER_PLANE_DIM_X,
            (y4-y3+N_THREADS_PER_PLANE_DIM_Y-1) / N_THREADS_PER_PLANE_DIM_Y);
        target_inner_3d_kernel<<<n_block_center, threadsPerBlock, 0, streams[0]>>>(nx,ny,nz,
            x3,x4,y3,y4,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_right(
            (z4-z3+N_THREADS_PER_PLANE_DIM_X-1) / N_THREADS_PER_PLANE_DIM_X,
            (y4-y3+N_THREADS_PER_PLANE_DIM_Y-1) / N_THREADS_PER_PLANE_DIM_Y,
            1);
        target_pml_3d_kernel<<<n_block_right, threadsPerBlock, 0, streams[4]>>>(nx,ny,nz,
            x5,x6,y3,y4,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_bottom(
            (z4-z3+N_THREADS_PER_PLANE_DIM_X-1) / N_THREADS_PER_PLANE_DIM_X,
            (y6-y5+N_THREADS_PER_PLANE_DIM_Y-1) / N_THREADS_PER_PLANE_DIM_Y,
            1);
        target_pml_3d_kernel<<<n_block_bottom, threadsPerBlock, 0, streams[5]>>>(nx,ny,nz,
            xmin,xmax,y5,y6,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_back(
            (z6-z5+N_THREADS_PER_PLANE_DIM_X-1) / N_THREADS_PER_PLANE_DIM_X,
            (ny+N_THREADS_PER_PLANE_DIM_Y-1) / N_THREADS_PER_PLANE_DIM_Y,
            1);
        target_pml_3d_kernel<<<n_block_back, threadsPerBlock, 0, streams[6]>>>(nx,ny,nz,
            xmin,xmax,ymin,ymax,z5,z6,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        for (int i = 0; i < num_streams; i++) {
            hipStreamSynchronize(streams[i]);
        }

        kernel_add_source_kernel<<<1, 1>>>(d_v, IDX3_l(sx,sy,sz), source[istep]);
        clock_gettime(CLOCK_REALTIME, &end);
        *time_kernel += (end.tv_sec  - start.tv_sec) +
                        (double)(end.tv_nsec - start.tv_nsec) / 1.0e9;

        float *t = d_u;
        d_u = d_v;
        d_v = t;

        // Print out
        if (istep % npo == 0) {
            printf("time step %u / %u\n", istep, nsteps);
        }
    }


    for (int i = 0; i < num_streams; i++) {
        hipStreamDestroy(streams[i]);
    }


    hipMemcpy(u, d_u, sizeof(float) * size_u, hipMemcpyDeviceToHost);

    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_vp);
    hipFree(d_phi);
    hipFree(d_eta);
}
