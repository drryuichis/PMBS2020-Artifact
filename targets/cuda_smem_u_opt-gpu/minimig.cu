#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <float.h>
#include <math.h>
#include <time.h>

#include "../../constants.h"

#define N_RADIUS 4
#define N_THREADS_PER_BLOCK_DIM 8

__global__ void target_inner_3d_kernel(
    llint nx, llint ny, llint nz,
    llint x3, llint x4, llint y3, llint y4, llint z3, llint z4,
    llint lx, llint ly, llint lz,
    float hdx_2, float hdy_2, float hdz_2,
    float coef0,
    float coefx_1, float coefx_2, float coefx_3, float coefx_4,
    float coefy_1, float coefy_2, float coefy_3, float coefy_4,
    float coefz_1, float coefz_2, float coefz_3, float coefz_4,
    const float *__restrict__ u, float *__restrict__ v, const float *__restrict__ vp,
    const float *__restrict__ phi, const float *__restrict__ eta
) {
    __shared__ float s_u[N_THREADS_PER_BLOCK_DIM+2*N_RADIUS][N_THREADS_PER_BLOCK_DIM+2*N_RADIUS][N_THREADS_PER_BLOCK_DIM+2*N_RADIUS];

    const llint i0 = x3 + blockIdx.z * blockDim.z;
    const llint j0 = y3 + blockIdx.y * blockDim.y;
    const llint k0 = z3 + blockIdx.x * blockDim.x;

    const llint i = i0 + threadIdx.z;
    const llint j = j0 + threadIdx.y;
    const llint k = k0 + threadIdx.x;

    const llint sui = threadIdx.z + N_RADIUS;
    const llint suj = threadIdx.y + N_RADIUS;
    const llint suk = threadIdx.x + N_RADIUS;

    const int z_side = threadIdx.z / N_RADIUS;
    s_u[threadIdx.z+z_side*N_THREADS_PER_BLOCK_DIM][suj][suk] = u[IDX3_l(i0+threadIdx.z+(z_side*2-1)*N_RADIUS,j,k)];
    const int y_side = threadIdx.y / N_RADIUS;
    s_u[sui][threadIdx.y+y_side*N_THREADS_PER_BLOCK_DIM][suk] = u[IDX3_l(i,j0+threadIdx.y+(y_side*2-1)*N_RADIUS,k)];
    s_u[sui][suj][threadIdx.x] = u[IDX3_l(i,j,k0+threadIdx.x-N_RADIUS)];
    s_u[sui][suj][threadIdx.x+N_THREADS_PER_BLOCK_DIM] = u[IDX3_l(i,j,k0+threadIdx.x+N_RADIUS)];

    __syncthreads();

    if (i > x4-1 || j > y4-1 || k > z4-1) { return; }

    float lap = __fmaf_rn(coef0, s_u[sui][suj][suk]
              , __fmaf_rn(coefx_1, __fadd_rn(s_u[sui+1][suj][suk],s_u[sui-1][suj][suk])
              , __fmaf_rn(coefy_1, __fadd_rn(s_u[sui][suj+1][suk],s_u[sui][suj-1][suk])
              , __fmaf_rn(coefz_1, __fadd_rn(s_u[sui][suj][suk+1],s_u[sui][suj][suk-1])
              , __fmaf_rn(coefx_2, __fadd_rn(s_u[sui+2][suj][suk],s_u[sui-2][suj][suk])
              , __fmaf_rn(coefy_2, __fadd_rn(s_u[sui][suj+2][suk],s_u[sui][suj-2][suk])
              , __fmaf_rn(coefz_2, __fadd_rn(s_u[sui][suj][suk+2],s_u[sui][suj][suk-2])
              , __fmaf_rn(coefx_3, __fadd_rn(s_u[sui+3][suj][suk],s_u[sui-3][suj][suk])
              , __fmaf_rn(coefy_3, __fadd_rn(s_u[sui][suj+3][suk],s_u[sui][suj-3][suk])
              , __fmaf_rn(coefz_3, __fadd_rn(s_u[sui][suj][suk+3],s_u[sui][suj][suk-3])
              , __fmaf_rn(coefx_4, __fadd_rn(s_u[sui+4][suj][suk],s_u[sui-4][suj][suk])
              , __fmaf_rn(coefy_4, __fadd_rn(s_u[sui][suj+4][suk],s_u[sui][suj-4][suk])
              , __fmul_rn(coefz_4, __fadd_rn(s_u[sui][suj][suk+4],s_u[sui][suj][suk-4])
    )))))))))))));

    v[IDX3_l(i,j,k)] = __fmaf_rn(2.f, s_u[sui][suj][suk],
        __fmaf_rn(vp[IDX3(i,j,k)], lap, -v[IDX3_l(i,j,k)])
    );
}

__global__ void target_pml_3d_kernel(
    llint nx, llint ny, llint nz,
    llint x3, llint x4, llint y3, llint y4, llint z3, llint z4,
    llint lx, llint ly, llint lz,
    float hdx_2, float hdy_2, float hdz_2,
    float coef0,
    float coefx_1, float coefx_2, float coefx_3, float coefx_4,
    float coefy_1, float coefy_2, float coefy_3, float coefy_4,
    float coefz_1, float coefz_2, float coefz_3, float coefz_4,
    const float *__restrict__ u, float *__restrict__ v, const float *__restrict__ vp,
    float *__restrict__ phi, const float *__restrict__ eta
) {
    __shared__ float s_u[N_THREADS_PER_BLOCK_DIM+2*N_RADIUS][N_THREADS_PER_BLOCK_DIM+2*N_RADIUS][N_THREADS_PER_BLOCK_DIM+2*N_RADIUS];

    const llint i0 = x3 + blockIdx.z * blockDim.z;
    const llint j0 = y3 + blockIdx.y * blockDim.y;
    const llint k0 = z3 + blockIdx.x * blockDim.x;

    const llint i = i0 + threadIdx.z;
    const llint j = j0 + threadIdx.y;
    const llint k = k0 + threadIdx.x;

    const llint sui = threadIdx.z + N_RADIUS;
    const llint suj = threadIdx.y + N_RADIUS;
    const llint suk = threadIdx.x + N_RADIUS;

    const int z_side = threadIdx.z / N_RADIUS;
    s_u[threadIdx.z+z_side*N_THREADS_PER_BLOCK_DIM][suj][suk] = u[IDX3_l(i0+threadIdx.z+(z_side*2-1)*N_RADIUS,j,k)];
    const int y_side = threadIdx.y / N_RADIUS;
    s_u[sui][threadIdx.y+y_side*N_THREADS_PER_BLOCK_DIM][suk] = u[IDX3_l(i,j0+threadIdx.y+(y_side*2-1)*N_RADIUS,k)];
    s_u[sui][suj][threadIdx.x] = u[IDX3_l(i,j,k0+threadIdx.x-N_RADIUS)];
    s_u[sui][suj][threadIdx.x+N_THREADS_PER_BLOCK_DIM] = u[IDX3_l(i,j,k0+threadIdx.x+N_RADIUS)];

    __syncthreads();

    if (i > x4-1 || j > y4-1 || k > z4-1) { return; }

    float lap = __fmaf_rn(coef0, s_u[sui][suj][suk]
        , __fmaf_rn(coefx_1, __fadd_rn(s_u[sui+1][suj][suk],s_u[sui-1][suj][suk])
        , __fmaf_rn(coefy_1, __fadd_rn(s_u[sui][suj+1][suk],s_u[sui][suj-1][suk])
        , __fmaf_rn(coefz_1, __fadd_rn(s_u[sui][suj][suk+1],s_u[sui][suj][suk-1])
        , __fmaf_rn(coefx_2, __fadd_rn(s_u[sui+2][suj][suk],s_u[sui-2][suj][suk])
        , __fmaf_rn(coefy_2, __fadd_rn(s_u[sui][suj+2][suk],s_u[sui][suj-2][suk])
        , __fmaf_rn(coefz_2, __fadd_rn(s_u[sui][suj][suk+2],s_u[sui][suj][suk-2])
        , __fmaf_rn(coefx_3, __fadd_rn(s_u[sui+3][suj][suk],s_u[sui-3][suj][suk])
        , __fmaf_rn(coefy_3, __fadd_rn(s_u[sui][suj+3][suk],s_u[sui][suj-3][suk])
        , __fmaf_rn(coefz_3, __fadd_rn(s_u[sui][suj][suk+3],s_u[sui][suj][suk-3])
        , __fmaf_rn(coefx_4, __fadd_rn(s_u[sui+4][suj][suk],s_u[sui-4][suj][suk])
        , __fmaf_rn(coefy_4, __fadd_rn(s_u[sui][suj+4][suk],s_u[sui][suj-4][suk])
        , __fmul_rn(coefz_4, __fadd_rn(s_u[sui][suj][suk+4],s_u[sui][suj][suk-4])
    )))))))))))));

    const float s_eta_c = eta[IDX3_eta1(i,j,k)];

    v[IDX3_l(i,j,k)] = __fdiv_rn(
        __fmaf_rn(
            __fmaf_rn(2.f, s_eta_c,
                __fsub_rn(2.f,
                    __fmul_rn(s_eta_c, s_eta_c)
                )
            ),
            s_u[sui][suj][suk],
            __fmaf_rn(
                vp[IDX3(i,j,k)],
                __fadd_rn(lap, phi[IDX3(i,j,k)]),
                -v[IDX3_l(i,j,k)]
            )
        ),
        __fmaf_rn(2.f, s_eta_c, 1.f)
    );

    phi[IDX3(i,j,k)] = __fdiv_rn(
            __fsub_rn(
                phi[IDX3(i,j,k)],
                __fmaf_rn(
                __fmul_rn(
                    __fsub_rn(eta[IDX3_eta1(i+1,j,k)], eta[IDX3_eta1(i-1,j,k)]),
                    __fsub_rn(s_u[sui+1][suj][suk], s_u[sui-1][suj][suk])
                ), hdx_2,
                __fmaf_rn(
                __fmul_rn(
                    __fsub_rn(eta[IDX3_eta1(i,j+1,k)], eta[IDX3_eta1(i,j-1,k)]),
                    __fsub_rn(s_u[sui][suj+1][suk], s_u[sui][suj-1][suk])
                ), hdy_2,
                __fmul_rn(
                    __fmul_rn(
                        __fsub_rn(eta[IDX3_eta1(i,j,k+1)], eta[IDX3_eta1(i,j,k-1)]),
                        __fsub_rn(s_u[sui][suj][suk+1], s_u[sui][suj][suk-1])
                    ),
                hdz_2)
                ))
            )
        ,
        __fadd_rn(1.f, s_eta_c)
    );
}

__global__ void kernel_add_source_kernel(float *g_u, llint idx, float source) {
    g_u[idx] += source;
}

extern "C" void target(
    uint nsteps, double *time_kernel,
    llint nx, llint ny, llint nz,
    llint x1, llint x2, llint x3, llint x4, llint x5, llint x6,
    llint y1, llint y2, llint y3, llint y4, llint y5, llint y6,
    llint z1, llint z2, llint z3, llint z4, llint z5, llint z6,
    llint lx, llint ly, llint lz,
    llint sx, llint sy, llint sz,
    float hdx_2, float hdy_2, float hdz_2,
    const float *__restrict__ coefx, const float *__restrict__ coefy, const float *__restrict__ coefz,
    float *__restrict__ u, const float *__restrict__ v, const float *__restrict__ vp,
    const float *__restrict__ phi, const float *__restrict__ eta, const float *__restrict__ source
) {
    struct timespec start, end;

    const llint size_u = (nx + 2 * lx) * (ny + 2 * ly) * (nz + 2 * lz);
    const llint size_v = size_u;
    const llint size_phi = nx*ny*nz;
    const llint size_vp = size_phi;
    const llint size_eta = (nx+2)*(ny+2)*(nz+2);

    const llint size_u_ext = ((((nx+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM + 1) * N_THREADS_PER_BLOCK_DIM) + 2 * lx)
                           * ((((ny+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM + 1) * N_THREADS_PER_BLOCK_DIM) + 2 * ly)
                           * ((((nz+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM + 1) * N_THREADS_PER_BLOCK_DIM) + 2 * lz);

    float *d_u, *d_v, *d_vp, *d_phi, *d_eta;
    hipMalloc(&d_u, sizeof(float) * size_u_ext);
    hipMalloc(&d_v, sizeof(float) * size_u_ext);
    hipMalloc(&d_vp, sizeof(float) * size_vp);
    hipMalloc(&d_phi, sizeof(float) * size_phi);
    hipMalloc(&d_eta, sizeof(float) * size_eta);

    hipMemcpy(d_u, u, sizeof(float) * size_u, hipMemcpyHostToDevice);
    hipMemcpy(d_v, v, sizeof(float) * size_v, hipMemcpyHostToDevice);
    hipMemcpy(d_vp, vp, sizeof(float) * size_vp, hipMemcpyHostToDevice);
    hipMemcpy(d_phi, phi, sizeof(float) * size_phi, hipMemcpyHostToDevice);
    hipMemcpy(d_eta, eta, sizeof(float) * size_eta, hipMemcpyHostToDevice);

    const llint xmin = 0; const llint xmax = nx;
    const llint ymin = 0; const llint ymax = ny;

    dim3 threadsPerBlock(N_THREADS_PER_BLOCK_DIM, N_THREADS_PER_BLOCK_DIM, N_THREADS_PER_BLOCK_DIM);

    int num_streams = 7;
    hipStream_t streams[num_streams];
    for (int i = 0; i < num_streams; i++) {
        hipStreamCreate(&(streams[i]));
    }

    const uint npo = 100;
    for (uint istep = 1; istep <= nsteps; ++istep) {
        clock_gettime(CLOCK_REALTIME, &start);

        dim3 n_block_front(
            (z2-z1+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (ny+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (nx+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM);
        target_pml_3d_kernel<<<n_block_front, threadsPerBlock, 0, streams[1]>>>(nx,ny,nz,
            xmin,xmax,ymin,ymax,z1,z2,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_top(
            (z4-z3+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (y2-y1+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (nx+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM);
        target_pml_3d_kernel<<<n_block_top, threadsPerBlock, 0, streams[2]>>>(nx,ny,nz,
            xmin,xmax,y1,y2,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_left(
            (z4-z3+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (y4-y3+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (x2-x1+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM);
        target_pml_3d_kernel<<<n_block_left, threadsPerBlock, 0, streams[3]>>>(nx,ny,nz,
            x1,x2,y3,y4,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_center(
            (z4-z3+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (y4-y3+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (x4-x3+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM);
        target_inner_3d_kernel<<<n_block_center, threadsPerBlock, 0, streams[0]>>>(nx,ny,nz,
            x3,x4,y3,y4,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_right(
            (z4-z3+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (y4-y3+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (x6-x5+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM);
        target_pml_3d_kernel<<<n_block_right, threadsPerBlock, 0, streams[4]>>>(nx,ny,nz,
            x5,x6,y3,y4,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_bottom(
            (z4-z3+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (y6-y5+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (nx+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM);
        target_pml_3d_kernel<<<n_block_bottom, threadsPerBlock, 0, streams[5]>>>(nx,ny,nz,
            xmin,xmax,y5,y6,z3,z4,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_back(
            (z6-z5+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (ny+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (nx+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM);
        target_pml_3d_kernel<<<n_block_back, threadsPerBlock, 0, streams[6]>>>(nx,ny,nz,
            xmin,xmax,ymin,ymax,z5,z6,
            lx,ly,lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        for (int i = 0; i < num_streams; i++) {
            hipStreamSynchronize(streams[i]);
        }

        kernel_add_source_kernel<<<1, 1>>>(d_v, IDX3_l(sx,sy,sz), source[istep]);
        clock_gettime(CLOCK_REALTIME, &end);
        *time_kernel += (end.tv_sec  - start.tv_sec) +
                        (double)(end.tv_nsec - start.tv_nsec) / 1.0e9;

        float *t = d_u;
        d_u = d_v;
        d_v = t;

        // Print out
        if (istep % npo == 0) {
            printf("time step %u / %u\n", istep, nsteps);
        }
    }


    for (int i = 0; i < num_streams; i++) {
        hipStreamDestroy(streams[i]);
    }


    hipMemcpy(u, d_u, sizeof(float) * size_u, hipMemcpyDeviceToHost);

    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_vp);
    hipFree(d_phi);
    hipFree(d_eta);
}

